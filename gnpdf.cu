#include "hip/hip_runtime.h"

#include "mex.h"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "npdf.cu"

#include <vector>
#include <map>
#include <set>

using namespace std;

#define MAXX 10
NPDF X[MAXX];

MTRand mt;
hiprandGenerator_t gen;
int initGPI = 0;

void GetString(const mxArray *string_array_ptr, char **buf) {
    size_t buflen;
    if ( mxIsChar(string_array_ptr) != 1)
        mexErrMsgTxt( "First input must be a string");
    if (mxGetM(string_array_ptr)!=1)
        mexErrMsgTxt( "First input must be a row vector");
    buflen = (mxGetM(string_array_ptr) * mxGetN(string_array_ptr)) + 1;
    *buf=(char*)mxCalloc(buflen, sizeof(char));
    *buf = mxArrayToString(string_array_ptr);
    //mexPrintf("%s\n",*buf);

}
//Input Check
int AddData(int nlhs,  mxArray *plhs[],int nrhs, const mxArray *prhs[], NPDF* Data) {
    const mwSize *dimsX,*dimsPi,*dimsSigma;
    mwSize  ndim; 
    if(nrhs!=6) {
        mexErrMsgTxt("action, data_set_index, X,Pi,mu,Sigma are needed.");
    }
    int dataset = (int)mxGetScalar(prhs[1]);
    if (dataset >= MAXX) {
        mexErrMsgTxt("data_set_index is too big.");
    }
    
    //check dimmension of x
    ndim = mxGetNumberOfDimensions(prhs[2]);
    if (ndim != 2) {
        mexErrMsgTxt("input data must be a 2D array");
    }    
    dimsX = mxGetDimensions(prhs[2]);
    Data[dataset].N = dimsX[0];
    Data[dataset].D = dimsX[1];
    
    //check dimension of pi
    ndim = mxGetNumberOfDimensions(prhs[3]);
    dimsPi = mxGetDimensions(prhs[3]);
    if (ndim==2 && (dimsPi[0]>1) && (dimsPi[1] > 1)) {
        mexErrMsgTxt("Pi must be a vector\n");
    }
    Data[dataset].T = mxGetNumberOfElements(prhs[3]); 
    
    //check mu
    ndim = mxGetNumberOfDimensions(prhs[4]);
    if (ndim != 2) {
        mexErrMsgTxt("mu must be a 2D array");
    }
    if (mxGetM(prhs[4]) != Data[dataset].T || mxGetN(prhs[4]) != Data[dataset].D) {
        mexErrMsgTxt("mu dimmensions do not match");
    }

    //check sigma
    ndim = mxGetNumberOfDimensions(prhs[5]);
    dimsSigma = mxGetDimensions(prhs[5]);
    if (ndim != 3) {
        mexErrMsgTxt("Sigma must be a 3D array");
    }
    if (dimsSigma[0] != Data[dataset].D || dimsSigma[1] !=Data[dataset].D || dimsSigma[2] != Data[dataset].T) {
        mexErrMsgTxt("Sigma dimmensions do not match");
    }
    return dataset;
}
int UpdateCluster(int nrhs, const mxArray *prhs[], NPDF* Data) {
    const mwSize *dimsPi,*dimsSigma;
    mwSize  ndim; 
    if(nrhs!=5) {
        mexErrMsgTxt("action, data_set_index, Pi,mu,Sigma are needed.");
    }
    int dataset = (int)mxGetScalar(prhs[1]);
    if (dataset >= MAXX) {
        mexErrMsgTxt("data_set_index is too big.");
    }

    //check dimension of pi
    ndim = mxGetNumberOfDimensions(prhs[2]);
    dimsPi = mxGetDimensions(prhs[2]);
    if (ndim==2 && (dimsPi[0]>1) && (dimsPi[1] > 1)) {
        mexErrMsgTxt("Pi must be a vector\n");
    }
    if (Data[dataset].T != mxGetNumberOfElements(prhs[2])) {
        mexErrMsgTxt("Pi does not match\n");
    }
    
    //check mu
    ndim = mxGetNumberOfDimensions(prhs[3]);
    if (ndim != 2) {
        mexErrMsgTxt("mu must be a 2D array");
    }
    if (mxGetM(prhs[3]) != Data[dataset].T || mxGetN(prhs[3]) != Data[dataset].D) {
        mexErrMsgTxt("mu dimmensions do not match");
    }

    //check sigma
    ndim = mxGetNumberOfDimensions(prhs[4]);
    dimsSigma = mxGetDimensions(prhs[4]);
    if (ndim != 3) {
        mexErrMsgTxt("Sigma must be a 3D array");
    }
    if (dimsSigma[0] != Data[dataset].D || dimsSigma[1] !=Data[dataset].D || dimsSigma[2] != Data[dataset].T) {
        mexErrMsgTxt("Sigma dimmensions do not match");
    }
    return dataset;
}


// ----------------- the MEX driver runs on the CPU --------------------
void mexFunction(int nlhs,  mxArray *plhs[], int nrhs, const mxArray *prhs[] ) {
    if( nrhs < 1 ) {
		mexErrMsgTxt( "Usage: at least one parameter is needed");
    }
    char *cmd = NULL;
    //char ** dataitems = NULL;
    GetString(prhs[0],&cmd);
    string strcmd(cmd);
    if (strcmd == "setdevice") {
        if (initGPI ==0) {
            if (nrhs!=3) { 
                mexErrMsgTxt( "Device and random seed are needed");
            }
            int device = (int)mxGetScalar(prhs[1]);
            int randseed = (int)mxGetScalar(prhs[2]);
            if (!X[0].SetDevice(device)) {
                mexErrMsgTxt( "Failed to set GPU device.");
            }
            mt.seed(randseed); 
            if (hiprandCreateGenerator(&gen,HIPRAND_RNG_PSEUDO_DEFAULT) != hipSuccess )  {
                mexErrMsgTxt( "Failed to initilize GPU random number generators.");
            }
            if (hiprandSetPseudoRandomGeneratorSeed(gen, randseed) != hipSuccess) {
                mexErrMsgTxt( "Failed to initilize GPU random number generators.");
            }
            initGPI = 1;
        } else {
            mexErrMsgTxt( "Device can only be set once");
        }
    } else if (strcmd == "clear") {
        if (initGPI > 0) {
            for (int i =0; i < MAXX; i++) {
                X[i].clear();
            }
            hiprandDestroyGenerator(gen);
        }
        initGPI = 0;
        
    } else if (strcmd == "adddata") {
        if (initGPI >0) {
            int dataset = AddData(nlhs,plhs,nrhs,prhs, X);    
            X[dataset].clear();
            X[dataset].getPaddedDim();
            X[dataset].AllocateHostMemory();

            double *iX = mxGetPr(prhs[2]);
            double *Pi = mxGetPr(prhs[3]);
            double *tMu = mxGetPr(prhs[4]);
            double *sigma = mxGetPr(prhs[5]);
            X[dataset].GetHostData(iX);
            X[dataset].GetHostData(Pi, tMu,sigma);
            X[dataset].InitGPU();
            X[dataset].CopyToGPU(1);
        } else {
            mexErrMsgTxt( "Device has to be set first");
        }
    } else if (strcmd == "updatecluster") {
        if (initGPI >0) {
            int dataset = UpdateCluster(nrhs,prhs, X);    
            double *Pi = mxGetPr(prhs[2]);
            double *tMu = mxGetPr(prhs[3]);
            double *sigma = mxGetPr(prhs[4]);
            X[dataset].GetHostData(Pi, tMu,sigma);
            X[dataset].CopyToGPU(0);
        } else {
            mexErrMsgTxt( "Device has to be set first");
        }
    } else if (strcmd == "pdf") {
        if (initGPI >0) {
            if (nrhs!=3) { 
                mexErrMsgTxt( "Usage:npdf('pdf',dataset,logscale)");
            }
            int dataset = (int)mxGetScalar(prhs[1]);
            int logscale = (int)mxGetScalar(prhs[2]);
            X[dataset].DoPDF(logscale,nlhs); 
            if (nlhs==1) {
                plhs[0]=mxCreateNumericMatrix(X[dataset].N, X[dataset].T, mxSINGLE_CLASS, mxREAL);
                float* r = (float*)mxGetData(plhs[0]);
                X[dataset].getDensity(r);
            }
        } else {
            mexErrMsgTxt( "Device has to be set first");
        }
    } else if (strcmd == "sum") {
        if (initGPI >0) {
            if (nrhs!=3) { 
                mexErrMsgTxt( "Usage:npdf('sum',dataset,Z)");
            }
            int dataset = (int)mxGetScalar(prhs[1]);
            double *Z = mxGetPr(prhs[2]);
            float result = X[dataset].Sum(Z);
            if (nlhs==1) {
                plhs[0]=mxCreateNumericMatrix(1, 1, mxDOUBLE_CLASS, mxREAL);
                double* r = (double*)mxGetData(plhs[0]);
                r[0] = result;
            }
        } else {
            mexErrMsgTxt( "Device has to be set first");
        }
    } else if (strcmd == "pdf&sum") {
        if (initGPI >0) {
            if (nrhs!=3) { 
                mexErrMsgTxt( "Usage:npdf('pdf',dataset,z4)");
            }
            int dataset = (int)mxGetScalar(prhs[1]);
            double *Z4 = mxGetPr(prhs[2]);
            X[dataset].DoPDF(0,0); 
            X[dataset].NormalizeDensity();
            float result = X[dataset].SumLog(Z4);
            if (nlhs==1) {
                plhs[0]=mxCreateNumericMatrix(1, 1, mxDOUBLE_CLASS, mxREAL);
                double* r = (double*)mxGetData(plhs[0]);
                r[0] = result;
            }
        } else {
            mexErrMsgTxt( "Device has to be set first");
        }
    } else if (strcmd == "density") {       //read computed density only
        if (initGPI >0) {
            if (nrhs!=2) { 
                mexErrMsgTxt( "Usage:gnpdf('density',dataset)");
            }
            int dataset = (int)mxGetScalar(prhs[1]);
            if (nlhs==1) {
                plhs[0]=mxCreateNumericMatrix(X[dataset].N, X[dataset].T, mxSINGLE_CLASS, mxREAL);
                float* r = (float*)mxGetData(plhs[0]);
                X[dataset].getDensity(r);
            }
        } else {
            mexErrMsgTxt( "Device has to be set first");
        }
    } else if (strcmd == "sample") {
        if (initGPI >0) {
            if (nrhs<3) { 
                mexErrMsgTxt( "Usage:npdf('sample',dataset,logscale,(optional)randomnumer)");
            }
            int dataset = (int)mxGetScalar(prhs[1]);
            int logscale = (int)mxGetScalar(prhs[2]);
            if (nrhs > 3) { //random number providedes
                //check random numbers
                mwSize ndim = mxGetNumberOfDimensions(prhs[3]);
                if (ndim==2 && (mxGetM(prhs[3])>1) && (mxGetN(prhs[3]) > 1)) {
                    mexErrMsgTxt("randomnumber must be a vector\n");
                }
                int temp = mxGetNumberOfElements(prhs[3]); 
                if (temp != X[dataset].N) {
                    mexErrMsgTxt("randomnumber dimmension does not match\n");
                }
                double *irn = mxGetPr(prhs[3]);
                X[dataset].GetRandNumber(irn);
            } else {
                X[dataset].GetRandNumber(gen);
            }
            X[dataset].DoSample(logscale); 
            if (nlhs==1) {
                plhs[0]=mxCreateNumericMatrix(X[dataset].N, 1, mxSINGLE_CLASS, mxREAL);
                float* r = (float*)mxGetData(plhs[0]);
                X[dataset].getIndicator(r);
            } 
        } else {
            mexErrMsgTxt( "Device has to be set first");
        }
    } else if (strcmd == "max") {
        if (initGPI >0) {
            if (nrhs<2) { 
                mexErrMsgTxt( "Usage:npdf('max',dataset");
            }
            int dataset = (int)mxGetScalar(prhs[1]);
            X[dataset].DoMax(); 
            if (nlhs==1) {
                plhs[0]=mxCreateNumericMatrix(X[dataset].N, 1, mxSINGLE_CLASS, mxREAL);
                float* r = (float*)mxGetData(plhs[0]);
                X[dataset].getIndicator(r);
            } 
        } else {
            mexErrMsgTxt( "Device has to be set first");
        }
    } else if (strcmd == "pdf&sample") {
        if (initGPI >0) {
            if (nrhs<2) { 
                mexErrMsgTxt( "Usage:npdf('pdf&sample',dataset,(optional)randomnumber)");
            }
            int dataset = (int)mxGetScalar(prhs[1]);
            if (nrhs > 2) { //random number providedes
                //check random numbers
                mwSize ndim = mxGetNumberOfDimensions(prhs[2]);
                if (ndim==2 && (mxGetM(prhs[2])>1) && (mxGetN(prhs[2]) > 1)) {
                    mexErrMsgTxt("randomnumber must be a vector\n");
                }
                int temp = mxGetNumberOfElements(prhs[2]); 
                if (temp != X[dataset].N) {
                    mexErrMsgTxt("randomnumber dimmension does not match\n");
                }
                double *irn = mxGetPr(prhs[2]);
                X[dataset].GetRandNumber(irn);
            } else {
                X[dataset].GetRandNumber(gen);
            }
            X[dataset].DoPDF(1,0); //logscale, no backtohost

            X[dataset].DoSample(1); 
            if (nlhs==1) {
                plhs[0]=mxCreateNumericMatrix(X[dataset].N, 1, mxSINGLE_CLASS, mxREAL);
                float* r = (float*)mxGetData(plhs[0]);
                X[dataset].getIndicator(r);
            }
        } else {
            mexErrMsgTxt( "Device has to be set first");
        }
    }  else if (strcmd == "pdf&sample-c") {
        if (initGPI >0) {
            if (nrhs<5) { 
                mexErrMsgTxt( "Usage:npdf('pdf&sample-c',dataset1,dataset2,Z,W,(optional)randomnumber)");
            }
            int dataset1 = (int)mxGetScalar(prhs[1]);
            int dataset2 = (int)mxGetScalar(prhs[2]);
            double *Z1 = mxGetPr(prhs[3]);
            double *wwk = mxGetPr(prhs[4]); 
            if (nrhs > 5) { //random number provideded
                double *irn = mxGetPr(prhs[5]);
                X[dataset2].GetRandNumber(irn);
            } else {
                X[dataset2].GetRandNumber(gen);
            }
            X[dataset2].DoPDFandSample_C(Z1,wwk,&X[dataset1]);
            if (nlhs==1) {
                plhs[0]=mxCreateNumericMatrix(X[dataset2].N, 1, mxSINGLE_CLASS, mxREAL);
                float* r = (float*)mxGetData(plhs[0]);
                X[dataset2].getIndicator(r);
            }
        } else {
            mexErrMsgTxt( "Device has to be set first");
        }
    } else if (strcmd == "zs") {
        if (initGPI >0) {
            if (nrhs<5) { 
                mexErrMsgTxt( "Usage:npdf('zs',dataset1,dataset2,W,Z4");
            }
            int dataset1 = (int)mxGetScalar(prhs[1]);
            int dataset2 = (int)mxGetScalar(prhs[2]);
            double *wwk = mxGetPr(prhs[3]); 
            double *Z4 = mxGetPr(prhs[4]);

            //update Z1
            X[dataset1].GetRandNumber(gen);
            X[dataset1].DoPDF(0,0); //logscale, no backtohost
            X[dataset1].NormalizeDensity();
            X[dataset1].DoSample(0); 
            plhs[0]=mxCreateNumericMatrix(X[dataset1].N, 1, mxSINGLE_CLASS, mxREAL);
            float* r = (float*)mxGetData(plhs[0]);
            X[dataset1].getIndicator(r);
            
            
            //update Z2 using old Z4
            X[dataset2].GetRandNumber(gen);
            X[dataset2].DoPDF(0,0); 
            X[dataset2].NormalizeDensity();
            X[dataset2].DoZ(Z4,wwk,&X[dataset1], 1);
            plhs[1]=mxCreateNumericMatrix(X[dataset2].N, 1, mxSINGLE_CLASS, mxREAL);
            r = (float*)mxGetData(plhs[1]);
            X[dataset2].getIndicator(r);

            //update Z4 using new Z2
            X[dataset1].GetRandNumber(gen);
            for (int i = 0; i <X[dataset1].N; i++) {
                Z4[i] = r[i];
            }
            X[dataset1].DoZ(Z4,wwk,&X[dataset2], 0);
            plhs[2]=mxCreateNumericMatrix(X[dataset1].N, 1, mxSINGLE_CLASS, mxREAL);
            r = (float*)mxGetData(plhs[2]);
            X[dataset1].getIndicator(r);
            

        } else {
            mexErrMsgTxt( "Device has to be set first");
        }
    } else if (strcmd == "zz") {
        if (initGPI >0) {
            if (nrhs<6) { 
                mexErrMsgTxt( "Usage:npdf('zz',dataset1,dataset2,W,Z5,what");
            }
            int dataset1 = (int)mxGetScalar(prhs[1]);
            int dataset2 = (int)mxGetScalar(prhs[2]);
            double *wwk = mxGetPr(prhs[3]); 
            double *Z5 = mxGetPr(prhs[4]);
            int what = (int)mxGetScalar(prhs[5]);

            //update Z1
            X[dataset1].GetRandNumber(gen);
            X[dataset1].DoPDF(0,0); //logscale, no backtohost
            X[dataset1].NormalizeDensity();
            if (what ==-1) {
                X[dataset1].DoMax(); 
            } else {
                X[dataset1].DoSample(0); 
            }
            plhs[0]=mxCreateNumericMatrix(X[dataset1].N, 1, mxSINGLE_CLASS, mxREAL);
            float* r = (float*)mxGetData(plhs[0]);
            X[dataset1].getIndicator(r);
            
            //update Z2 using Z5
            X[dataset1].DoZ_max_no_sampling(Z5,wwk,&X[dataset2], 0);
            plhs[1]=mxCreateNumericMatrix(X[dataset1].N, 1, mxSINGLE_CLASS, mxREAL);
            r = (float*)mxGetData(plhs[1]);
            X[dataset1].getIndicator(r);
        } else {
            mexErrMsgTxt( "Device has to be set first");
        }
    } else if (strcmd == "z5") {
        if (initGPI >0) {
            if (nrhs<5) { 
                mexErrMsgTxt( "Usage:npdf('z5',dataset1,dataset2,W,action");
            }
            int dataset1 = (int)mxGetScalar(prhs[1]);
            int dataset2 = (int)mxGetScalar(prhs[2]);
            double *wwk = mxGetPr(prhs[3]); 
            int what = (int)mxGetScalar(prhs[4]);
            X[dataset2].GetRandNumber(gen);
            X[dataset2].DoZ5(wwk,&X[dataset1],what);
            if (what <=0) { //-1 for max, 0 for density
                plhs[0]=mxCreateNumericMatrix(X[dataset2].N, 1, mxSINGLE_CLASS, mxREAL);
                float *r = (float*)mxGetData(plhs[0]);
                X[dataset2].getIndicator(r);                
            } else {
                if (nlhs > 0) {
                    plhs[0]=mxCreateNumericMatrix(X[dataset2].N, X[dataset2].T, mxSINGLE_CLASS, mxREAL);
                    float* r = (float*)mxGetData(plhs[0]);
                    X[dataset2].getDensity(r);
                }
            }
        } else {
            mexErrMsgTxt( "Device has to be set first");
        }
    } else if (strcmd == "qq") {
        if (initGPI >0) {
            int dataset1 = (int)mxGetScalar(prhs[1]);
            int dataset2 = (int)mxGetScalar(prhs[2]);
            if (nrhs ==6) { //need to reavluate the pdf
                //check indicators 
                int temp = mxGetNumberOfElements(prhs[3]); 
                if (temp != X[dataset1].T) {
                    mexErrMsgTxt("indicator dimmension does not match\n");
                }
                double *weight = mxGetPr(prhs[3]);
                X[dataset1].UpdateWeight(weight);
                X[dataset1].CopyToGPU(0);
                double *wwk = mxGetPr(prhs[4]); 
                double *Z2 = mxGetPr(prhs[5]); 
                X[dataset2].DoAltQQ(wwk, &X[dataset1],Z2);
                
            } else {
                mexErrMsgTxt( "Usage:npdf('qq',dataset1,dataset2,Weight_alt,WWK0,Z2)");
            }
            
            if (nlhs==1) {
                plhs[0]=mxCreateNumericMatrix(1, 1, mxDOUBLE_CLASS, mxREAL);
                double* r = mxGetPr(plhs[0]);
                r[0] = X[dataset2].QQ;
            }
        } else {
            mexErrMsgTxt( "Device has to be set first");
        }
    }else if (strcmd == "z1z2") {
        if (initGPI >0) {
            if (nrhs<4) { 
                mexErrMsgTxt( "Usage:npdf('z1z2',dataset1,dataset2,wwk0,(optional)randomnumber1,(optional)randomnumber2)");
            }
            int dataset1 = (int)mxGetScalar(prhs[1]);
            int dataset2 = (int)mxGetScalar(prhs[2]);
            if (nrhs > 4) { //random number providedes
                //check random numbers1
                mwSize ndim = mxGetNumberOfDimensions(prhs[4]);
                if (ndim==2 && (mxGetM(prhs[4])>1) && (mxGetN(prhs[4]) > 1)) {
                    mexErrMsgTxt("randomnumber1 must be a vector\n");
                }
                int temp = mxGetNumberOfElements(prhs[4]); 
                if (temp != X[dataset1].N) {
                    mexErrMsgTxt("randomnumber1 dimmension does not match\n");
                }
                double *irn = mxGetPr(prhs[4]);
                X[dataset1].GetRandNumber(irn);

                //check random numbers1
                ndim = mxGetNumberOfDimensions(prhs[5]);
                if (ndim==2 && (mxGetM(prhs[5])>1) && (mxGetN(prhs[5]) > 1)) {
                    mexErrMsgTxt("randomnumber2 must be a vector\n");
                }
                temp = mxGetNumberOfElements(prhs[5]); 
                if (temp != X[dataset2].N) {
                    mexErrMsgTxt("randomnumber2 dimmension does not match\n");
                }
                irn = mxGetPr(prhs[5]);
                X[dataset2].GetRandNumber(irn);

            } else {
                X[dataset1].GetRandNumber(gen);
                X[dataset2].GetRandNumber(gen);
            }
            
            double *w = mxGetPr(prhs[3]);  
            if (!X[dataset2].DoZ1Z2(w,&X[dataset1])) {
                 mexErrMsgTxt("Allocating device memory failed\n");
            }
            
            if (nlhs>=1) {
                plhs[0]=mxCreateNumericMatrix(X[dataset1].N, 1, mxSINGLE_CLASS, mxREAL);
                float* r = (float*)mxGetData(plhs[0]);
                X[dataset1].getIndicator(r);
            }
            if (nlhs>=2) {
                plhs[1]=mxCreateNumericMatrix(X[dataset2].N, 1, mxSINGLE_CLASS, mxREAL);
                float* r = (float*)mxGetData(plhs[1]);
                X[dataset2].getIndicator(r);
            }
            if (nlhs>=3) {
                plhs[2]=mxCreateNumericMatrix(1, 1, mxDOUBLE_CLASS, mxREAL);
                double* r = mxGetPr(plhs[2]);
                r[0] = X[dataset2].QQ;
            }
        } else {
            mexErrMsgTxt( "Device has to be set first");
        }
    } 
}